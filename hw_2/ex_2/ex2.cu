#include "hip/hip_runtime.h"

#include <stdio.h>
#include <sys/time.h>
#include <cmath>

// Define a small relative error threshold
const double relativeErrorThreshold = 1e-6;

#define DataType double

// Compute C = A * B
__global__ void gemm(DataType *A, DataType *B, DataType *C, int numARows,
                      int numAColumns, int numBRows, int numBColumns){
  int r = blockIdx.x*blockDim.x+threadIdx.x;
  int c = blockIdx.y*blockDim.y+threadIdx.y;
  //@@ Insert code to implement matrix multiplication here

  if (r < numARows && c < numBColumns) {
    DataType tmp = 0;
    for (int i = 0; i < numBRows; i++) {
        tmp += A[r * numAColumns + i] * B[i * numBColumns + c];
    }
    C[numBColumns * r + c] = tmp;
  }
  
   
}

int main(int argc, char **argv) {
  
  DataType *hostA; // The A matrix
  DataType *hostB; // The B matrix
  DataType *hostC; // The output C matrix
  DataType *resultRef; // The reference result
  DataType *deviceA;
  DataType *deviceB;
  DataType *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;
  int numCColumns;

  //@@ Insert code below to read in numARows, numAColumns, numBColumns from args
  numARows = atoi(argv[2]);
  numAColumns = atoi(argv[3]);
  numBRows = atoi(argv[4]);
  numBColumns = atoi(argv[5]);
  numCRows = atoi(argv[6]);
  numCColumns = atoi(argv[7]);
  printf("Input matrix dim (%d x %d) (%d x %d) (%d x %d)\n", numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
  
  //@@ Insert code below to allocate Host memory for input and output
  hostA = (DataType *) malloc(numARows * numAColumns * sizeof(DataType));
  hostB = (DataType *) malloc(numBRows * numBColumns * sizeof(DataType));
  hostC = (DataType *) malloc(numCRows * numCColumns * sizeof(DataType));
  resultRef = (DataType *) malloc(numCRows * numCColumns * sizeof(DataType));
  
  //@@ Insert code below to initialize hostA and hostB to random numbers, and create reference result in CPU
  srand(time(0));  
  for (int i=0; i<numARows; i++){
    for (int j=0; j<numAColumns; j++){
      hostA[i*numAColumns+j] = rand();
    }     
  }
  for (int i=0; i<numBRows; i++){
    for (int j=0; j<numBColumns; j++){
      hostB[i*numBColumns+j] = rand();
    }     
  }
  for (int i=0; i<numCRows; i++){
    for (int j=0; j<numCColumns; j++){
      double tmp = 0;
      for (int k=0; k<numBRows; k++){
        tmp += hostA[i*numAColumns+k]*hostB[k*numBColumns+j];
      }
      resultRef[i*numCColumns+j] = tmp;
    }     
  }

  //@@ Insert code below to allocate GPU memory here
  hipMalloc(&deviceA, numARows*numAColumns*sizeof(DataType));
  hipMalloc(&deviceB, numBRows*numBColumns*sizeof(DataType));
  hipMalloc(&deviceC, numCRows*numCColumns*sizeof(DataType));

  
  //@@ Insert code to below to Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, numARows*numAColumns*sizeof(DataType), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows*numBColumns*sizeof(DataType), hipMemcpyHostToDevice);
  hipMemcpy(deviceC, hostC, numCRows*numCColumns*sizeof(DataType), hipMemcpyHostToDevice);


  //@@ Initialize the grid and block dimensions here
  dim3 TPB(atoi(argv[1]), atoi(argv[1]));
  int n = (numCRows*numCColumns + atoi(argv[1]) - 1)/atoi(argv[1]);
  dim3 BPG((numARows + atoi(argv[1]) - 1) / atoi(argv[1]), (numBColumns + atoi(argv[1]) - 1) / atoi(argv[1]));
  //dim3 BPG(n, n);
  
  //@@ Launch the GPU Kernel here
  gemm<<<BPG, TPB>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns);


  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, numCRows*numCColumns*sizeof(DataType), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  //@@ Insert code below to compare the output with the reference
  int equal = 1;
  for (int i=0; i<numCRows; i++){
    for (int j=0; j<numCColumns; j++){
        double absDiff = fabs(resultRef[i * numCColumns + j] - hostC[i * numCColumns + j]);
        double absResult = fabs(resultRef[i * numCColumns + j]);
        double relativeError = absDiff / (absResult == 0.0 ? 1.0 : absResult);

        if (relativeError > relativeErrorThreshold) {
            printf("The results are not equal (%d,%d). ", i, j);
            printf("resultRef: %lf, hostC: %lf\n", resultRef[i * numCColumns + j], hostC[i * numCColumns + j]);
            equal = 0;
        }
    }     
  }
  if (equal){
    printf("The output of host are equal to reference\n");
  }


  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  //@@ Free the CPU memory here
  free(hostA);
  free(hostB);
  free(hostC);
  free(resultRef);

  return 0;
}
